#include "hip/hip_runtime.h"
#include "cuda_half.h"

#include <memory>
#include <string>
#include <utility>
#include "exr.h"
#include "image.h"
#include "timer.h"

#include "blur.cu"  // Include source directly for now

struct CudaFreeFunctor {
  void operator()(void* p) noexcept { cudaCheckError(hipFree(p)); }
};

template <class T>
std::unique_ptr<T, CudaFreeFunctor> cuda_malloc_unique(size_t size) {
  T* device_image;
  cudaCheckError(hipMalloc(&device_image, size));
  return std::unique_ptr<T, CudaFreeFunctor>(device_image);
}

template <class T>
auto alloc_and_copy(T* host_image, image_dims dims) {
  using deviceT = typename std::remove_const<T>::type;
  auto size = allocated_bytes(dims);
  auto device_image = cuda_malloc_unique<deviceT>(size);
  cudaCheckError(
      hipMemcpy(device_image.get(), host_image, size, hipMemcpyDefault));
  return device_image;
}

void copy_image(void* dest, const void* source, image_dims dims) {
  cudaCheckError(
      hipMemcpy(dest, source, allocated_bytes(dims), hipMemcpyDefault));
}

// Reduce width but keep stride unchanged, to test handling of padded images.
// Fill padding with infinite values, which should make it obvious if these
// pixels are accidentally accessed.
void crop_image(float4* image, image_dims& dims) {
  dims.width -= 9;
  auto inf = std::numeric_limits<float>::infinity();

  for (size_t y = 0; y < dims.height; y++) {
    for (size_t x = dims.width; x < dims.stride_pixels; x++) {
      image[pixel_index(dims, x, y)] = {inf, inf, inf, inf};
    }
  }
}

const bool do_crop = false;
const bool do_texture = false;

int main(int argc, char** argv) {
  int radius = (argc > 3) ? std::stoi(argv[3]) : 5;
  int n_passes = (argc > 4) ? std::stoi(argv[4]) : 3;

  auto [pixels, _dims] = read_exr(argv[1]);
  // Work around stupid "structured binding can't be captured" issue
  auto dims = _dims;

  if (do_crop)
    crop_image(pixels.get(), dims);

  auto source = alloc_and_copy(pixels.get(), dims);
  auto dest = cuda_malloc_unique<float4>(allocated_bytes(dims));
  auto temp = cuda_malloc_unique<float4>(allocated_bytes(dims));

  timeit("smooth blur", [&]() {
    smooth_blur(dest.get(), source.get(), temp.get(), dims, radius, n_passes);
  });

  if (do_texture) {
    timeit("texture blur", [&]() {
      smooth_blur_texture(dest.get(), source.get(), temp.get(), dims, radius,
                          n_passes);
    });
  }

  // Copy result back to host and write
  copy_image(pixels.get(), dest.get(), dims);
  write_exr(argv[2], dims, pixels.get());

  return 0;
}
