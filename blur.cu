#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"

#define cudaCheckError(code)                                         \
  {                                                                  \
    if ((code) != hipSuccess) {                                     \
      handleCudaError(__FILE__, __LINE__, hipGetErrorString(code)); \
    }                                                                \
  }

void handleCudaError(const char* file, int line, const char* error) {
  std::cerr << "CUDA failure " << file << ":" << line << " " << error << "\n";
}

static int n_blocks(int threads, int block_size) {
  return (threads + block_size - 1) / block_size;
}

// ===== Operators for built-in vector types =====
__device__ static float4 operator*(const float4& a, const float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}

__device__ static float4& operator+=(float4& a, const float4& b) {
  a = {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
  return a;
}

__device__ static float4& operator-=(float4& a, const float4& b) {
  a = {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
  return a;
}

// ===== General Utilities =====

// Type used for image data. This will eventually be templated to support
// different vector lengths, as well as half-float data.
using ImageT = float4;
// Type of temporary values. This will always be some kind of float, even
// when ImageT is half-float.
using TempT = float4;

__device__ static int cuda_index() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

static hipTextureObject_t create_texture(const ImageT* source,
                                          image_dims dims) {
  hipTextureDesc tex_desc = {};
  tex_desc.addressMode[0] = hipAddressModeClamp;
  tex_desc.addressMode[1] = hipAddressModeClamp;
  tex_desc.filterMode = hipFilterModePoint;
  tex_desc.readMode = hipReadModeElementType;
  tex_desc.normalizedCoords = 0;

  hipResourceDesc resource_desc = {hipResourceTypePitch2D};
  resource_desc.res.pitch2D.devPtr = const_cast<ImageT*>(source);
  resource_desc.res.pitch2D.desc = hipCreateChannelDesc<ImageT>();
  resource_desc.res.pitch2D.width = dims.width;
  resource_desc.res.pitch2D.height = dims.height;
  resource_desc.res.pitch2D.pitchInBytes = stride_bytes(dims);

  hipTextureObject_t tex = 0;
  cudaCheckError(
      hipCreateTextureObject(&tex, &resource_desc, &tex_desc, nullptr));
  return tex;
}

// ===== Transpose =====

// Naive transpose
// Supporting vector types with a shared-memory transpose is non-trivial, and
// this is fast enough for now.
__global__ void transpose_kernel(ImageT* dest, const ImageT* source,
                                 image_dims dims) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dims.width || y >= dims.height)
    return;

  dest[x * dims.height + y] = source[pixel_index(dims, x, y)];
}

void transpose(ImageT* dest, const ImageT* source, image_dims dims) {
  dim3 grid(n_blocks(dims.width, 16), n_blocks(dims.height, 16));
  dim3 threads(16, 16);
  transpose_kernel<<<grid, threads>>>(dest, source, dims);
}

// ===== Blur functions =====
__global__ void vertical_box_blur_kernel(ImageT* dest, const ImageT* source,
                                         image_dims dims, int radius) {
  int x = cuda_index();
  if (x >= dims.width)
    return;

  float scale = 1.0f / (2 * radius + 1);

  // Fill initial box, repeating the edge pixel
  TempT edge = source[pixel_index(dims, x, 0)];
  TempT sum = edge * (radius + 1);
  for (int y = 1; y < radius + 1; y++) {
    sum += source[pixel_index(dims, x, y)];
  }

  // Compute result pixels
  int top = -radius;
  int bottom = radius;
  for (int y = 0; y < dims.height; y++) {
    dest[pixel_index(dims, x, y)] = sum * scale;

    // Shift the box
    sum -= source[pixel_index(dims, x, max(top, 0))];
    top++;
    bottom++;
    sum += source[pixel_index(dims, x, min(bottom, int(dims.height - 1)))];
  }
}

// Horizontal box blur using texture. This is significantly slower (approx
// 2x) than transposing and using the vertical blur.
__global__ void horizontal_box_blur_kernel(ImageT* dest,
                                           hipTextureObject_t source,
                                           image_dims dims, int radius) {
  int y = cuda_index();
  if (y >= dims.height)
    return;

  float scale = 1.0f / (2 * radius + 1);

  // Fill initial box, repeating the edge pixel
  TempT edge = tex2D<ImageT>(source, 0, y);
  TempT sum = edge * (radius + 1);
  for (int x = 1; x < radius + 1; x++) {
    sum += tex2D<ImageT>(source, x, y);
  }

  // Compute result pixels
  int left = -radius;
  int right = radius;
  for (int x = 0; x < dims.width; x++) {
    dest[pixel_index(dims, x, y)] = sum * scale;

    // Shift the box
    sum -= tex2D<ImageT>(source, left, y);
    left++;
    right++;
    sum += tex2D<ImageT>(source, right, y);
  }
}

void vertical_box_blur(ImageT* dest, const ImageT* source, image_dims dims,
                       int radius) {
  const int BLOCK_SIZE = 128;
  int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
  vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(dest, source, dims,
                                                     radius);
}

void box_blur(ImageT* dest, const ImageT* source, ImageT* temp, image_dims dims,
              int radius) {
  const int BLOCK_SIZE = 128;

  // Vertical blur
  {
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, source, dims,
                                                       radius);
  }

  transpose(dest, temp, dims);

  // Horizontal blur
  {
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    int grid_dim = n_blocks(transpose_dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, dest,
                                                       transpose_dims, radius);
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(dest, temp,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
}

// Blur an image with a repeated box blur.
// 2 passes approximates a triangle filter, 3 is close enough to a Gaussian.
// The radius param indicates the total radius: it will be divided
// more-or-less equally between the passes.
// dest and source may point to the same image (if you don't need to keep the
// source image and want to save some memory).
void smooth_blur(ImageT* dest, const ImageT* source, ImageT* temp,
                 image_dims dims, int radius, int n_passes) {
  const int BLOCK_SIZE = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Vertical blur
  {
    int remaining = radius;
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(to, s, dims,
                                                         this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Horizontal blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    int grid_dim = n_blocks(transpose_dims.width, BLOCK_SIZE);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Like smooth_blur, but use textures for horizontal blur
void smooth_blur_texture(ImageT* dest, const ImageT* source, ImageT* temp,
                         image_dims dims, int radius, int n_passes) {
  const int BLOCK_SIZE = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Vertical blur
  {
    int remaining = radius;
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(to, s, dims,
                                                         this_radius);
      std::swap(to, from);
    }
  }

  // Horizontal blur
  {
    int remaining = radius;
    int grid_dim = n_blocks(dims.height, BLOCK_SIZE);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;

      if (false) {
        hipMemcpy(to, from, allocated_bytes(dims), hipMemcpyDefault);
      } else {
        auto tex = create_texture(from, dims);
        horizontal_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(to, tex, dims,
                                                             this_radius);
        cudaCheckError(hipDestroyTextureObject(tex));
      }

      std::swap(to, from);
    }
  }
}
