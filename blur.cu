#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"

#define cudaCheckError(code)                                         \
  {                                                                  \
    if ((code) != hipSuccess) {                                     \
      handleCudaError(__FILE__, __LINE__, hipGetErrorString(code)); \
    }                                                                \
  }

void handleCudaError(const char* file, int line, const char* error) {
  std::cerr << "CUDA failure " << file << ":" << line << " " << error << "\n";
}

static int n_blocks(int threads, int block_size) {
  return (threads + block_size - 1) / block_size;
}

// ===== Operators for built-in vector types =====
__device__ static float4 operator*(const float4& a, const float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}

__device__ static float4& operator+=(float4& a, const float4& b) {
  a = {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
  return a;
}

__device__ static float4& operator-=(float4& a, const float4& b) {
  a = {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
  return a;
}

// ===== General Utilities =====

// Type used for image data. This will eventually be templated to support
// different vector lengths, as well as half-float data.
using ImageT = float4;
// Type of temporary values. This will always be some kind of float, even
// when ImageT is half-float.
using TempT = float4;

// Naive transpose
// Supporting vector types with a shared-memory transpose is non-trivial, and
// this is fast enough for now.
__global__ void transpose_kernel(ImageT* dest, const ImageT* source,
                                 image_dims dims) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dims.width || y >= dims.height)
    return;

  dest[x * dims.height + y] = source[pixel_index(dims, x, y)];
}

void transpose(ImageT* dest, const ImageT* source, image_dims dims) {
  dim3 grid(n_blocks(dims.width, 16), n_blocks(dims.height, 16));
  dim3 threads(16, 16);
  transpose_kernel<<<grid, threads>>>(dest, source, dims);
}

// ===== Blur functions =====
__device__ static int cuda_index() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void vertical_box_blur_kernel(ImageT* dest, const ImageT* source,
                                         image_dims dims, int radius) {
  int x = cuda_index();
  if (x >= dims.width)
    return;

  float scale = 1.0f / (2 * radius + 1);

  // Fill initial box, repeating the edge pixel
  TempT edge = source[pixel_index(dims, x, 0)];
  TempT sum = edge * (radius + 1);
  for (int y = 1; y < radius + 1; y++) {
    sum += source[pixel_index(dims, x, y)];
  }

  // Compute result pixels
  int top = -radius;
  int bottom = radius;
  for (int y = 0; y < dims.height; y++) {
    dest[pixel_index(dims, x, y)] = sum * scale;

    // Shift the box
    sum -= source[pixel_index(dims, x, max(top, 0))];
    top++;
    bottom++;
    sum += source[pixel_index(dims, x, min(bottom, int(dims.height - 1)))];
  }
}

void vertical_box_blur(ImageT* dest, const ImageT* source, image_dims dims,
                       int radius) {
  const int BLOCK_SIZE = 128;
  int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
  vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(dest, source, dims,
                                                     radius);
}
