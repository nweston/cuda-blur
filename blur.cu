#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"

// ===== Pixel functions =====

// All of the blur kernels have two variables: ImageT which is the type
// stored in global memory, and TempT which is used for thread-local
// values. ImageT is given as a template argument, and TempT is derived from
// ImageT via the temp_type template.

// To add a new image type, you must specialize temp_type, to_temp and
// from_temp accordingly.

// Temp types have to be something we can do math on, which usually means a
// float or float vector. Specifically they must support += and -= (with the
// same type on the rhs), and multiplication by a float. You must also
// specialize black() for each temp type.

// For example, when ImageT=half2, the corresponding TempT is float2. For
// floating point types, ImageT and TempT will be the same.
//

// Type of temporary values. This will always be some kind of float, even
// when ImageT is half-float.
template <class ImageT>
struct temp_type {
  using type = ImageT;
};

template <>
struct temp_type<half2> {
  using type = float2;
};

// Convert T to its corresponding TempT
template <class T>
__device__ static typename temp_type<T>::type to_temp(const T& a) {
  return a;
}
template <>
float2 to_temp(const half2& a) {
  return {a.x, a.y};
}

// Convert to T from its corresponding TempT
template <class T>
__device__ static T from_temp(const typename temp_type<T>::type& a) {
  return a;
}
template <>
half2 from_temp(const float2& a) {
  return {a.x, a.y};
}

// Return a single pixel value with all channels set to black.
template <class T>
__device__ static T black(){};

template <>
__device__ float4 black<float4>() {
  return {0, 0, 0, 0};
}
template <>
__device__ float2 black<float2>() {
  return {0, 0};
}
template <>
__device__ float black<float>() {
  return 0;
}

/// float4 ///
__device__ static float4 operator*(const float4& a, const float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}

__device__ static float4& operator+=(float4& a, const float4& b) {
  a = {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
  return a;
}

__device__ static float4& operator-=(float4& a, const float4& b) {
  a = {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
  return a;
}

/// float2 ///
__device__ static float2 operator*(const float2& a, const float b) {
  return {a.x * b, a.y * b};
}

__device__ static float2& operator+=(float2& a, const float2& b) {
  a = {a.x + b.x, a.y + b.y};
  return a;
}

__device__ static float2& operator-=(float2& a, const float2& b) {
  a = {a.x - b.x, a.y - b.y};
  return a;
}
// ===== General Utilities =====

#define cudaCheckError(code)                                           \
  {                                                                    \
    if ((code) != hipSuccess) {                                       \
      handle_cuda_error(__FILE__, __LINE__, hipGetErrorString(code)); \
    }                                                                  \
  }

void default_error_handler(const char* file, int line, const char* error) {
  std::cerr << "CUDA failure " << file << ":" << line << " " << error << "\n";
}

// Called whenever a CUDA function returns an error. Reassign to provide your
// own error handling.
using error_handler_t = void (*)(const char*, int, const char*);
error_handler_t handle_cuda_error = &default_error_handler;

static int n_blocks(int threads, int block_size, int outputs_per_thread = 1) {
  int block_outputs = block_size * outputs_per_thread;
  return (threads + block_outputs - 1) / block_outputs;
}

__device__ static int cuda_index_x() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ static int cuda_index_y() {
  return blockIdx.y * blockDim.y + threadIdx.y;
}

template <class ImageT>
__device__ static typename temp_type<ImageT>::type get_pixel(
    const ImageT* image, const image_dims& dims, int x, int y) {
  return to_temp(image[pixel_index(dims, x, y)]);
}

template <class ImageT>
__device__ static void set_pixel(
    ImageT* image, const image_dims& dims, int x, int y,
    const typename temp_type<ImageT>::type& value) {
  image[pixel_index(dims, x, y)] = from_temp<ImageT>(value);
}

// ===== Transpose =====

// Naive transpose
// Supporting vector types with a shared-memory transpose is non-trivial, and
// this is fast enough for now.
template <class ImageT>
__global__ void transpose_kernel(ImageT* dest, const ImageT* source,
                                 image_dims dims) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dims.width || y >= dims.height)
    return;

  dest[x * dims.height + y] = source[pixel_index(dims, x, y)];
}

template <class ImageT>
void transpose(ImageT* dest, const ImageT* source, image_dims dims) {
  dim3 grid(n_blocks(dims.width, 16), n_blocks(dims.height, 16));
  dim3 threads(16, 16);
  transpose_kernel<<<grid, threads>>>(dest, source, dims);
}

// ===== Blur functions =====
template <class ImageT>
__global__ void vertical_box_blur_kernel(ImageT* dest, const ImageT* source,
                                         image_dims dims, int radius) {
  using TempT = typename temp_type<ImageT>::type;

  // Split each column into vertical slices
  int n_slices = blockDim.y;
  int slice_height = (dims.height + n_slices - 1) / n_slices;
  int y_start = threadIdx.y * slice_height;
  int y_limit = min(static_cast<int>(dims.height), y_start + slice_height);

  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    TempT sum;

    // Fill initial box, repeating the edge pixel
    if (y_start == 0) {
      TempT edge = get_pixel(source, dims, x, 0);
      sum = edge * (radius + 1);
      for (int y = 1; y < radius + 1; y++) {
        sum += get_pixel(source, dims, x, y);
      }
    } else {
      sum = black<TempT>();
      for (int y = y_start - radius; y < y_start + radius + 1; y++) {
        sum += get_pixel(source, dims, x, max(y, 0));
      }
    }

    // Compute result pixels
    int top = y_start - radius;
    int bottom = y_start + radius;
    for (int y = y_start; y < y_limit; y++) {
      set_pixel(dest, dims, x, y, sum * scale);

      // Shift the box
      sum -= get_pixel(source, dims, x, max(top, 0));
      top++;
      bottom++;
      sum += get_pixel(source, dims, x, min(bottom, int(dims.height - 1)));
    }
  }
}

// Vertical box blur, implemented by direct convolution instead of sliding
// window method.
template <class ImageT>
__global__ void vertical_direct_box_blur_kernel(ImageT* dest,
                                                const ImageT* source,
                                                image_dims dims, int radius) {
  int y = cuda_index_y();
  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    auto sum = get_pixel(source, dims, x, y);
    for (int yi = y - radius; yi < y; yi++) {
      sum += get_pixel(source, dims, x, max(yi, 0));
    }
    for (int yi = y + 1; yi < y + radius + 1; yi++) {
      sum += get_pixel(source, dims, x, min(yi, int(dims.height - 1)));
    }

    set_pixel(dest, dims, x, y, sum * scale);
  }
}

// Horizontal box blur, implemented by direct convolution instead of sliding
// window method.
template <class ImageT>
__global__ void horizontal_direct_box_blur_kernel(ImageT* dest,
                                                  const ImageT* source,
                                                  image_dims dims, int radius) {
  int y = cuda_index_y();
  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    auto sum = get_pixel(source, dims, x, y);
    for (int xi = x - radius; xi < x; xi++) {
      sum += get_pixel(source, dims, max(xi, 0), y);
    }
    for (int xi = x + 1; xi < x + radius + 1; xi++) {
      sum += get_pixel(source, dims, min(xi, int(dims.width - 1)), y);
    }

    set_pixel(dest, dims, x, y, sum * scale);
  }
}

#include "weights.h"

template <class ImageT>
__global__ void horizontal_precomputed_gaussian_blur_kernel(
    ImageT* dest, const ImageT* source, image_dims dims, int radius) {
  int y = cuda_index_y();

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    auto sum = get_pixel(source, dims, x, y) * weights[radius][0];
    for (int xi = x - radius; xi < x; xi++) {
      int dx = x - xi;
      float w = weights[radius][dx];
      sum += get_pixel(source, dims, max(xi, 0), y) * w;
    }
    for (int xi = x + 1; xi < x + radius + 1; xi++) {
      int dx = xi - x;
      float w = weights[radius][dx];
      sum += get_pixel(source, dims, min(xi, int(dims.width - 1)), y) * w;
    }

    set_pixel(dest, dims, x, y, sum);
  }
}

template <class ImageT>
__global__ void vertical_precomputed_gaussian_blur_kernel(ImageT* dest,
                                                          const ImageT* source,
                                                          image_dims dims,
                                                          int radius) {
  int y = cuda_index_y();

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    auto sum = get_pixel(source, dims, x, y) * weights[radius][0];
    for (int yi = y - radius; yi < y; yi++) {
      int dy = y - yi;
      float w = weights[radius][dy];
      sum += get_pixel(source, dims, x, max(yi, 0)) * w;
    }
    for (int yi = y + 1; yi < y + radius + 1; yi++) {
      int dy = yi - y;
      float w = weights[radius][dy];
      sum += get_pixel(source, dims, x, min(yi, int(dims.height - 1))) * w;
    }

    set_pixel(dest, dims, x, y, sum);
  }
}

// Blur an image with a repeated box blur.
// 2 passes approximates a triangle filter, 3 is close enough to a Gaussian.
// The radius param indicates the total radius: it will be divided
// more-or-less equally between the passes.
// dest and source may point to the same image (if you don't need to keep the
// source image and want to save some memory).
template <class ImageT>
void smooth_blur(ImageT* dest, const ImageT* source, ImageT* temp,
                 image_dims dims, int radius, int n_passes,
                 int outputs_per_thread_v = 1, int outputs_per_thread_h = 1,
                 int threads_per_column_v = 1, int threads_per_column_h = 1) {
  const int BLOCK_WIDTH = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  // Vertical blur
  {
    int remaining = radius;
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_v);
    int grid_dim = n_blocks(dims.width, BLOCK_WIDTH, outputs_per_thread_v);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(to, s, dims,
                                                        this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Horizontal blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_h);
    int grid_dim =
        n_blocks(transpose_dims.width, BLOCK_WIDTH, outputs_per_thread_h);
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Use horizontal and vertical direct blur kernels, without transposing.
template <class ImageT>
void direct_blur_no_transpose(ImageT* dest, const ImageT* source, ImageT* temp,
                              image_dims dims, int radius, int n_passes,
                              int outputs_per_thread = 1) {
  dim3 BLOCK_DIM(16, 8);

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  dim3 grid_dim(n_blocks(dims.width, BLOCK_DIM.x, outputs_per_thread),
                n_blocks(dims.height, BLOCK_DIM.y));

  // Horizontal blur
  {
    int remaining = radius;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      horizontal_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, s, dims,
                                                                 this_radius);
      std::swap(to, from);
    }
  }

  // Vertical blur
  {
    int remaining = radius;

    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, from, dims,
                                                               this_radius);
      std::swap(to, from);
    }
  }

  assert(from == dest);
}

template <class ImageT>
void precomputed_gaussian_blur(ImageT* dest, const ImageT* source, ImageT* temp,
                               image_dims dims, int radius,
                               int outputs_per_thread = 1) {
  dim3 BLOCK_DIM(16, 8);
  dim3 grid_dim(n_blocks(dims.width, BLOCK_DIM.x, outputs_per_thread),
                n_blocks(dims.height, BLOCK_DIM.y));

  assert(radius <= MAX_PRECOMPUTED_RADIUS);

  horizontal_precomputed_gaussian_blur_kernel<<<grid_dim, BLOCK_DIM>>>(
      temp, source, dims, radius);
  vertical_precomputed_gaussian_blur_kernel<<<grid_dim, BLOCK_DIM>>>(
      dest, temp, dims, radius);
}
