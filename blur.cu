#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"

#define cudaCheckError(code)                                         \
  {                                                                  \
    if ((code) != hipSuccess) {                                     \
      handleCudaError(__FILE__, __LINE__, hipGetErrorString(code)); \
    }                                                                \
  }

void handleCudaError(const char* file, int line, const char* error) {
  std::cerr << "CUDA failure " << file << ":" << line << " " << error << "\n";
}

static int n_blocks(int threads, int block_size) {
  return (threads + block_size - 1) / block_size;
}

// ===== Operators for built-in vector types =====
__device__ static float4 operator*(const float4& a, const float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}

__device__ static float4& operator+=(float4& a, const float4& b) {
  a = {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
  return a;
}

__device__ static float4& operator-=(float4& a, const float4& b) {
  a = {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
  return a;
}

// ===== General Utilities =====

// Type used for image data. This will eventually be templated to support
// different vector lengths, as well as half-float data.
using ImageT = float4;
// Type of temporary values. This will always be some kind of float, even
// when ImageT is half-float.
using TempT = float4;

__device__ static int cuda_index() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

static hipTextureObject_t create_texture(const ImageT* source,
                                          image_dims dims) {
  hipTextureDesc tex_desc = {};
  tex_desc.addressMode[0] = hipAddressModeClamp;
  tex_desc.addressMode[1] = hipAddressModeClamp;
  tex_desc.filterMode = hipFilterModePoint;
  tex_desc.readMode = hipReadModeElementType;
  tex_desc.normalizedCoords = 0;

  hipResourceDesc resource_desc = {hipResourceTypePitch2D};
  resource_desc.res.pitch2D.devPtr = const_cast<ImageT*>(source);
  resource_desc.res.pitch2D.desc = hipCreateChannelDesc<ImageT>();
  resource_desc.res.pitch2D.width = dims.width;
  resource_desc.res.pitch2D.height = dims.height;
  resource_desc.res.pitch2D.pitchInBytes = stride_bytes(dims);

  hipTextureObject_t tex = 0;
  cudaCheckError(
      hipCreateTextureObject(&tex, &resource_desc, &tex_desc, nullptr));
  return tex;
}

// ===== Transpose =====

// Naive transpose
// Supporting vector types with a shared-memory transpose is non-trivial, and
// this is fast enough for now.
__global__ void transpose_kernel(ImageT* dest, const ImageT* source,
                                 image_dims dims) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dims.width || y >= dims.height)
    return;

  dest[x * dims.height + y] = source[pixel_index(dims, x, y)];
}

void transpose(ImageT* dest, const ImageT* source, image_dims dims) {
  dim3 grid(n_blocks(dims.width, 16), n_blocks(dims.height, 16));
  dim3 threads(16, 16);
  transpose_kernel<<<grid, threads>>>(dest, source, dims);
}

// ===== Blur functions =====
__global__ void vertical_box_blur_kernel(ImageT* dest, const ImageT* source,
                                         image_dims dims, int radius) {
  // Split each column into vertical slices
  int n_slices = blockDim.y;
  int slice_height = (dims.height + n_slices - 1) / n_slices;
  int y_start = threadIdx.y * slice_height;
  int y_limit = min(static_cast<int>(dims.height), y_start + slice_height);

  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index(); x < dims.width; x += blockDim.x * gridDim.x) {
    TempT sum;

    // Fill initial box, repeating the edge pixel
    if (y_start == 0) {
      TempT edge = source[pixel_index(dims, x, 0)];
      sum = edge * (radius + 1);
      for (int y = 1; y < radius + 1; y++) {
        sum += source[pixel_index(dims, x, y)];
      }
    } else {
      sum = {0, 0, 0, 0};
      for (int y = y_start - radius; y < y_start + radius + 1; y++) {
        sum += source[pixel_index(dims, x, max(y, 0))];
      }
    }

    // Compute result pixels
    int top = y_start - radius;
    int bottom = y_start + radius;
    for (int y = y_start; y < y_limit; y++) {
      dest[pixel_index(dims, x, y)] = sum * scale;

      // Shift the box
      sum -= source[pixel_index(dims, x, max(top, 0))];
      top++;
      bottom++;
      sum += source[pixel_index(dims, x, min(bottom, int(dims.height - 1)))];
    }
  }
}

// Horizontal box blur using texture. This is significantly slower (approx
// 2x) than transposing and using the vertical blur.
__global__ void horizontal_box_blur_kernel(ImageT* dest,
                                           hipTextureObject_t source,
                                           image_dims dims, int radius) {
  int y = cuda_index();
  if (y >= dims.height)
    return;

  float scale = 1.0f / (2 * radius + 1);

  // Fill initial box, repeating the edge pixel
  TempT edge = tex2D<ImageT>(source, 0, y);
  TempT sum = edge * (radius + 1);
  for (int x = 1; x < radius + 1; x++) {
    sum += tex2D<ImageT>(source, x, y);
  }

  // Compute result pixels
  int left = -radius;
  int right = radius;
  for (int x = 0; x < dims.width; x++) {
    dest[pixel_index(dims, x, y)] = sum * scale;

    // Shift the box
    sum -= tex2D<ImageT>(source, left, y);
    left++;
    right++;
    sum += tex2D<ImageT>(source, right, y);
  }
}

void vertical_box_blur(ImageT* dest, const ImageT* source, image_dims dims,
                       int radius) {
  const int BLOCK_SIZE = 128;
  int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
  vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(dest, source, dims,
                                                     radius);
}

void box_blur(ImageT* dest, const ImageT* source, ImageT* temp, image_dims dims,
              int radius) {
  const int BLOCK_SIZE = 128;

  // Vertical blur
  {
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, source, dims,
                                                       radius);
  }

  transpose(dest, temp, dims);
  image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                               dims.sizeof_channel, dims.height};

  // Horizontal blur
  {
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    int grid_dim = n_blocks(transpose_dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, dest,
                                                       transpose_dims, radius);
  }

  // Transpose back to the original format.
  transpose(dest, temp, transpose_dims);
}

// Blur an image with a repeated box blur.
// 2 passes approximates a triangle filter, 3 is close enough to a Gaussian.
// The radius param indicates the total radius: it will be divided
// more-or-less equally between the passes.
// dest and source may point to the same image (if you don't need to keep the
// source image and want to save some memory).
void smooth_blur(ImageT* dest, const ImageT* source, ImageT* temp,
                 image_dims dims, int radius, int n_passes,
                 int outputs_per_thread_v = 1, int outputs_per_thread_h = 1,
                 int threads_per_column_v = 1, int threads_per_column_h = 1) {
  const int BLOCK_WIDTH = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Vertical blur
  {
    int remaining = radius;
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_v);
    int grid_dim = n_blocks(dims.width, BLOCK_WIDTH) / outputs_per_thread_v;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(to, s, dims,
                                                        this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Horizontal blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_h);
    int grid_dim =
        n_blocks(transpose_dims.width, BLOCK_WIDTH) / outputs_per_thread_h;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Like smooth_blur, but use textures for horizontal blur
void smooth_blur_texture(ImageT* dest, const ImageT* source, ImageT* temp,
                         image_dims dims, int radius, int n_passes,
                         int outputs_per_thread_v = 1,
                         int outputs_per_thread_h = 1) {
  const int BLOCK_SIZE = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Vertical blur
  {
    int remaining = radius;
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE) / outputs_per_thread_v;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(to, s, dims,
                                                         this_radius);
      std::swap(to, from);
    }
  }

  // Horizontal blur
  {
    int remaining = radius;
    int grid_dim = n_blocks(dims.height, BLOCK_SIZE) / outputs_per_thread_h;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;

      if (false) {
        hipMemcpy(to, from, allocated_bytes(dims), hipMemcpyDefault);
      } else {
        auto tex = create_texture(from, dims);
        horizontal_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(to, tex, dims,
                                                             this_radius);
        cudaCheckError(hipDestroyTextureObject(tex));
      }

      std::swap(to, from);
    }
  }
}
