#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"

#define cudaCheckError(code)                                         \
  {                                                                  \
    if ((code) != hipSuccess) {                                     \
      handleCudaError(__FILE__, __LINE__, hipGetErrorString(code)); \
    }                                                                \
  }

void handleCudaError(const char* file, int line, const char* error) {
  std::cerr << "CUDA failure " << file << ":" << line << " " << error << "\n";
}

static int n_blocks(int threads, int block_size) {
  return (threads + block_size - 1) / block_size;
}

// ===== Operators for built-in vector types =====
__device__ static float4 operator*(const float4& a, const float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}

__device__ static float4& operator+=(float4& a, const float4& b) {
  a = {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
  return a;
}

__device__ static float4& operator-=(float4& a, const float4& b) {
  a = {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
  return a;
}

// ===== General Utilities =====

// Type used for image data. This will eventually be templated to support
// different vector lengths, as well as half-float data.
using ImageT = float4;
// Type of temporary values. This will always be some kind of float, even
// when ImageT is half-float.
using TempT = float4;

__device__ static int cuda_index_x() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ static int cuda_index_y() {
  return blockIdx.y * blockDim.y + threadIdx.y;
}

// ===== Transpose =====

// Naive transpose
// Supporting vector types with a shared-memory transpose is non-trivial, and
// this is fast enough for now.
__global__ void transpose_kernel(ImageT* dest, const ImageT* source,
                                 image_dims dims) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dims.width || y >= dims.height)
    return;

  dest[x * dims.height + y] = source[pixel_index(dims, x, y)];
}

void transpose(ImageT* dest, const ImageT* source, image_dims dims) {
  dim3 grid(n_blocks(dims.width, 16), n_blocks(dims.height, 16));
  dim3 threads(16, 16);
  transpose_kernel<<<grid, threads>>>(dest, source, dims);
}

// ===== Blur functions =====
__global__ void vertical_box_blur_kernel(ImageT* dest, const ImageT* source,
                                         image_dims dims, int radius) {
  // Split each column into vertical slices
  int n_slices = blockDim.y;
  int slice_height = (dims.height + n_slices - 1) / n_slices;
  int y_start = threadIdx.y * slice_height;
  int y_limit = min(static_cast<int>(dims.height), y_start + slice_height);

  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    TempT sum;

    // Fill initial box, repeating the edge pixel
    if (y_start == 0) {
      TempT edge = source[pixel_index(dims, x, 0)];
      sum = edge * (radius + 1);
      for (int y = 1; y < radius + 1; y++) {
        sum += source[pixel_index(dims, x, y)];
      }
    } else {
      sum = {0, 0, 0, 0};
      for (int y = y_start - radius; y < y_start + radius + 1; y++) {
        sum += source[pixel_index(dims, x, max(y, 0))];
      }
    }

    // Compute result pixels
    int top = y_start - radius;
    int bottom = y_start + radius;
    for (int y = y_start; y < y_limit; y++) {
      dest[pixel_index(dims, x, y)] = sum * scale;

      // Shift the box
      sum -= source[pixel_index(dims, x, max(top, 0))];
      top++;
      bottom++;
      sum += source[pixel_index(dims, x, min(bottom, int(dims.height - 1)))];
    }
  }
}

// Vertical box blur, implemented by direct convolution instead of sliding
// window method.
__global__ void vertical_direct_box_blur_kernel(ImageT* dest,
                                                const ImageT* source,
                                                image_dims dims, int radius) {
  int y = cuda_index_y();
  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    TempT sum = source[pixel_index(dims, x, y)];
    for (int yi = y - radius; yi < y; yi++) {
      sum += source[pixel_index(dims, x, max(yi, 0))];
    }
    for (int yi = y + 1; yi < y + radius + 1; yi++) {
      sum += source[pixel_index(dims, x, min(yi, int(dims.height - 1)))];
    }

    dest[pixel_index(dims, x, y)] = sum * scale;
  }
}

// Horizontal box blur, implemented by direct convolution instead of sliding
// window method.
__global__ void horizontal_direct_box_blur_kernel(ImageT* dest,
                                                  const ImageT* source,
                                                  image_dims dims, int radius) {
  int y = cuda_index_y();
  float scale = 1.0f / (2 * radius + 1);

  for (int x = cuda_index_x(); x < dims.width; x += blockDim.x * gridDim.x) {
    TempT sum = source[pixel_index(dims, x, y)];
    for (int xi = x - radius; xi < x; xi++) {
      sum += source[pixel_index(dims, max(xi, 0), y)];
    }
    for (int xi = x + 1; xi < x + radius + 1; xi++) {
      sum += source[pixel_index(dims, min(xi, int(dims.width - 1)), y)];
    }

    dest[pixel_index(dims, x, y)] = sum * scale;
  }
}

void vertical_box_blur(ImageT* dest, const ImageT* source, image_dims dims,
                       int radius) {
  const int BLOCK_SIZE = 128;
  int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
  vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(dest, source, dims,
                                                     radius);
}

void box_blur(ImageT* dest, const ImageT* source, ImageT* temp, image_dims dims,
              int radius) {
  const int BLOCK_SIZE = 128;

  // Vertical blur
  {
    int grid_dim = n_blocks(dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, source, dims,
                                                       radius);
  }

  transpose(dest, temp, dims);
  image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                               dims.sizeof_channel, dims.height};

  // Horizontal blur
  {
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    int grid_dim = n_blocks(transpose_dims.width, BLOCK_SIZE);
    vertical_box_blur_kernel<<<grid_dim, BLOCK_SIZE>>>(temp, dest,
                                                       transpose_dims, radius);
  }

  // Transpose back to the original format.
  transpose(dest, temp, transpose_dims);
}

// Blur an image with a repeated box blur.
// 2 passes approximates a triangle filter, 3 is close enough to a Gaussian.
// The radius param indicates the total radius: it will be divided
// more-or-less equally between the passes.
// dest and source may point to the same image (if you don't need to keep the
// source image and want to save some memory).
void smooth_blur(ImageT* dest, const ImageT* source, ImageT* temp,
                 image_dims dims, int radius, int n_passes,
                 int outputs_per_thread_v = 1, int outputs_per_thread_h = 1,
                 int threads_per_column_v = 1, int threads_per_column_h = 1) {
  const int BLOCK_WIDTH = 128;

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  // Vertical blur
  {
    int remaining = radius;
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_v);
    int grid_dim = n_blocks(dims.width, BLOCK_WIDTH) / outputs_per_thread_v;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(to, s, dims,
                                                        this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Horizontal blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    dim3 block_dim(BLOCK_WIDTH, threads_per_column_h);
    int grid_dim =
        n_blocks(transpose_dims.width, BLOCK_WIDTH) / outputs_per_thread_h;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_box_blur_kernel<<<grid_dim, block_dim>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Use vertical box blurs with direct convolution
void direct_blur_vertical(ImageT* dest, const ImageT* source, ImageT* temp,
                          image_dims dims, int radius, int n_passes,
                          int outputs_per_thread = 1) {
  dim3 BLOCK_DIM(16, 8);

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  // Vertical blur
  {
    int remaining = radius;
    dim3 grid_dim(n_blocks(dims.width, BLOCK_DIM.x) / outputs_per_thread,
                  n_blocks(dims.height, BLOCK_DIM.y));
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      vertical_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, s, dims,
                                                               this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Horizontal blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    dim3 grid_dim(
        n_blocks(transpose_dims.width, BLOCK_DIM.x) / outputs_per_thread,
        n_blocks(transpose_dims.height, BLOCK_DIM.y));

    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Use horizontal box blurs with direct convolution
void direct_blur_horizontal(ImageT* dest, const ImageT* source, ImageT* temp,
                            image_dims dims, int radius, int n_passes,
                            int outputs_per_thread = 1) {
  dim3 BLOCK_DIM(16, 8);

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  // Horizontal blur
  {
    int remaining = radius;
    dim3 grid_dim(n_blocks(dims.width, BLOCK_DIM.x) / outputs_per_thread,
                  n_blocks(dims.height, BLOCK_DIM.y));
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      horizontal_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, s, dims,
                                                                 this_radius);
      std::swap(to, from);
    }
  }

  transpose(to, from, dims);
  std::swap(to, from);

  // Vertical blur
  {
    int remaining = radius;
    // Transpose turns any horizontal padding into vertical padding. Ignore
    // those extra pixels when blurring.
    image_dims transpose_dims = {dims.height, dims.width, dims.channel_count,
                                 dims.sizeof_channel, dims.height};
    dim3 grid_dim(
        n_blocks(transpose_dims.width, BLOCK_DIM.x) / outputs_per_thread,
        n_blocks(transpose_dims.height, BLOCK_DIM.y));

    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      horizontal_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(
          to, from, transpose_dims, this_radius);
      std::swap(to, from);
    }
  }

  // Transpose back to the original format. This version of the dims includes
  // the extra height.
  transpose(to, from,
            {dims.height, dims.stride_pixels, dims.channel_count,
             dims.sizeof_channel, dims.height});
  assert(to == dest);
}

// Use horizontal and vertical direct blur kernels, without transposing.
void direct_blur_no_transpose(ImageT* dest, const ImageT* source, ImageT* temp,
                              image_dims dims, int radius, int n_passes,
                              int outputs_per_thread = 1) {
  dim3 BLOCK_DIM(16, 8);

  // Each blur pass and transpose needs to read from one image and write to
  // another. To avoid any copying, we swap these pointers around after each
  // operation.
  ImageT* from = dest;
  ImageT* to = temp;

  // Ensure that each pass will have radius >= 1. This speeds up small blurs
  // by reducing the number of passes.
  n_passes = std::min(n_passes, radius);

  dim3 grid_dim(n_blocks(dims.width, BLOCK_DIM.x) / outputs_per_thread,
                n_blocks(dims.height, BLOCK_DIM.y));

  // Horizontal blur
  {
    int remaining = radius;
    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      // On the very first pass, read from the source image
      const ImageT* s = (i == 0) ? source : from;
      horizontal_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, s, dims,
                                                                 this_radius);
      std::swap(to, from);
    }
  }

  // Vertical blur
  {
    int remaining = radius;

    for (int i = 0; i < n_passes; i++) {
      int this_radius = remaining / (n_passes - i);
      remaining -= this_radius;
      vertical_direct_box_blur_kernel<<<grid_dim, BLOCK_DIM>>>(to, from, dims,
                                                               this_radius);
      std::swap(to, from);
    }
  }

  assert(from == dest);
}
